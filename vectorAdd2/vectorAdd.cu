
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    x[i] = x[i] + i;
}
 
int main(void)
{
  int N = 2000;
  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
  }
 
  // Launch kernel on 1M elements on the GPU
  int blockSize = 32;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x);
  // for(int j = 0; j < 100000; j++)
   for (int i = 0; i < N; i++) {
    y[i] = 2.0f;
  }
    add<<<numBlocks, blockSize>>>(N, y);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = fmax(maxError, fabs(y[i]-3.0f));
//   std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}