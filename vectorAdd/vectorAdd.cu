
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += 1)
    x[i] = x[i] + i;
}
 
int main(void)
{
  int N = 16*1024+8*1024;;
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
  }
  int blockSize = 32;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x);
  for(int j = 0; j < 100000; j++)
   for (int i = 0; i < N; i++) {
    y[i] = 2.0f;
  }
  add<<<numBlocks, blockSize>>>(N, y);


  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = fmax(maxError, fabs(y[i]-3.0f));
//   std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}